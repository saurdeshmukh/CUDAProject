#include "hip/hip_runtime.h"
#include "quicksort.h"
#include<hip/hip_runtime.h>

__device__ void swap(Point2D *a,Point2D* b)
{
Point2D t; 
t.x = a->x;
t.y=a->y;
a->x= b->x;
a->y=b->y;
b->x = t.x;
b->y=t.y;
}

__device__ int partition(Point2D  *data,int l,int r,int x)
{
 Point2D pivot = data [r];
 int i = l-1;

 for(int j = l;j<=r-1;j++)
 {
   if(x==1)
   {
   	if(data[j].x <= pivot.x)
    	{
    	 i++;
     	swap(&data[i],&data[j]); 
     	}
   }
   else
   {
	if(data[j].y <= pivot.y)
        {
         i++;
        swap(&data[i],&data[j]);
        }
    }
 }
 swap(&data[i+1],&data[r]);
 return (i+1);
}

__global__ void qsort(Point2D *data,int  l,int r,int x)
{

if(l<r)
{
int pivot = partition (data,l,r,x);

hipStream_t s1,s2;
unsigned int flag = hipStreamDefault;

hipStreamCreateWithFlags(&s1,flag);
hipStreamCreateWithFlags(&s2,flag);

qsort<<<1,1,0,s1>>>(data,l,pivot-1,x);
qsort<<<1,1,0,s1>>>(data,pivot+1,r,x);
}
}


