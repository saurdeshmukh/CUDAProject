#include <stdio.h>
#include<limits.h>
#include <hip/hip_runtime.h>
#define maxWIDTH 350
#define BOUND 100
#include<time.h>

typedef struct
{
int x;
int y;
}Point2D;

__host__ void getRandomInputPoints(int n,Point2D * ptr)
{
        srand(0);
        int i=0;
        int pointx=0,pointy=0;
        for(i=0;i<n;i++)
        {
                pointx=rand()%BOUND;
                ptr->x=pointx;
                pointy=rand()%BOUND;
                ptr->y=pointy;
                ptr++;
        }
}

__host__ void printArray(Point2D*ptr,int n)
{
    int i=0;
    for(i=0;i<n;i++)
    {
        printf("Point %d - X: %d , Y: %d\n",i+1,ptr[i].x,ptr[i].y);

    }

}
__device__ int  minAreaFunction(int *ptr)
{
	int minArea=INT_MAX;
	for(int i=0;i<(maxWIDTH*maxWIDTH);i++)
		{
			if(ptr[i]<minArea && ptr[i]!=0)
			{
				minArea=ptr[i];
			}
		}
	return minArea;

}
__device__ void sortedArea(int *area,int *currentPoint)
{       
        int totalpoint=*currentPoint;
        
        int i,j,temp;
        for(i=0;i<totalpoint-1;i++)
        {       
                for(j=0;j<totalpoint-i-1;j++)
                {       
                        if(area[j]>area[j+1])
                        {       
                                temp=area[j+1];
                                area[j+1]=area[j];
                                area[j]=temp;
                        }
                }
        
        }
}


__device__ int * getArea(Point2D *ptr,int len,int leftEdge,int bottomEdge,int *currentPoint)
{
        int totalpoint=*currentPoint;
        int *area=(int*)malloc(totalpoint * sizeof(int));
        int i=0;
        int ar=0;

        for(i=0;i<totalpoint;i++)
        {
                if((ptr[i].x+1-leftEdge) > (ptr[i].y+1-bottomEdge))
                {
                        ar=(int)(ptr[i].x+1-leftEdge)*((ptr[i].x+1-leftEdge));
                }
                else
                {
                        ar=(int)(ptr[i].y+1-bottomEdge)*((ptr[i].y+1-bottomEdge));
                }
                area[i]=ar;
        }
        return area;
}

__device__ Point2D* getAboveRightPoints(int startx,int starty,Point2D* ptrx,Point2D* ptry,int len,int *currentPoint)
{
    int k=0;
    int i,j;
    int count=0;
    int totalpoint=*currentPoint;
    Point2D* temp;
    int check=0;
    for(i=startx;i<len;i++)
    {
        for(j=starty;j<len;j++)
        {
            if(ptrx[i].x==ptry[j].x && ptrx[i].y==ptry[j].y)
            {
                count++;
		break;
            }
        }
    }
    
    totalpoint=count;
    
    
    temp=(Point2D*)malloc(totalpoint*sizeof(Point2D));
    for(j=starty;j<len;j++)
    {
        for(i=startx;i<len;i++)
        {
            if(ptry[j].x==ptrx[i].x && ptry[j].y==ptrx[i].y)
            {
		check=0;
		for(int m=0;m<k;m++)
			if(ptry[j].x==temp[m].x && ptry[j].y==temp[m].y)
				check=1;
		if(check==0)
		{
                	temp[k].x=ptry[j].x;
                	temp[k].y=ptry[j].y;
                	k++;
			break;
		}
            }
        }
    }
	
    *currentPoint=k;
    return temp;
}

__device__ void sortingbyAxis(Point2D *points,Point2D *sorted,int n,int x)
{

	int i,j;
	Point2D temp;
        for(i=0;i<n;i++)
        {        
                sorted[i].x=points[i].x;
                sorted[i].y=points[i].y;
        }
        //sort by x co-ordinate
        if(x==1)
        {       
                for(i=0;i<n-1;i++)
                        {
                        for(j=0;j<n-i-1;j++)
                                {       
                                        if(sorted[j].x>sorted[j+1].x)
                                                {       
                                                        temp.x=sorted[j].x;
                                                        temp.y=sorted[j].y;
                                                        sorted[j].x=sorted[j+1].x;
                                                        sorted[j].y=sorted[j+1].y;
                                                        sorted[j+1].x=temp.x;
                                                        sorted[j+1].y=temp.y;
                                                }
                                        }
                                }
        }
	//sort by y co-ordinate
        else
        {       
                for(i=0;i<n-1;i++)
                        {
                        for(j=0;j<n-i-1;j++)
                                {       
                                        if(sorted[j].y>sorted[j+1].y)
                                                {       
                                                        temp.x=sorted[j].x;
                                                        temp.y=sorted[j].y;
                                                        sorted[j].x=sorted[j+1].x;
                                                        sorted[j].y=sorted[j+1].y;
                                                        sorted[j+1].x=temp.x;
                                                        sorted[j+1].y=temp.y;
                                                }
                                        }
                                }
        }

}
__global__ void k_bounding_algorithm(Point2D * points,int n,int k,int *finalArea,Point2D * sortedX,Point2D * sortedY,int *minArea)
{
	int threadId = blockDim.x*blockIdx.x + threadIdx.x;
	if(threadId < ((n-k+1)*(n-k+1)))
	{
	int i=threadId / (n-k+1);
	int j=threadId %(n-k+1);
	Point2D bottomPoint;
        Point2D leftPoint;
        Point2D *Rpoints=NULL;
        int *area;
	int totalpoints=0;
        minArea[threadId]=0;
	if(threadId==0)
	{
		sortingbyAxis(points,sortedX,n,1);
		sortingbyAxis(points,sortedY,n,0);
	}
	minArea[threadId]=INT_MAX;
	__syncthreads();
	leftPoint=sortedX[i];
        bottomPoint=sortedY[j];
	if(leftPoint.x <= bottomPoint.x && leftPoint.y >= bottomPoint.y)
	{
            int leftEdge=(int)(leftPoint.x-1);
            int bottomEdge=(int)(bottomPoint.y-1);
            Rpoints=getAboveRightPoints(i,j,sortedX,sortedY,n,&totalpoints);
		if(totalpoints>=k)
                {
			area=getArea(Rpoints,n,leftEdge,bottomEdge,&totalpoints);
            		sortedArea(area,&totalpoints);
			minArea[threadId]=area[k-1];
		}
	}
	__syncthreads();
	if(threadId ==0)
        {
                *finalArea=minAreaFunction(minArea);
		free(Rpoints);
		free(area);
        }
    }

}

int main(void)
{
    hipError_t err = hipSuccess;
    hipEvent_t seq_start,seq_stop;    

   int n=0;
   int k=0;
    
    printf("\nEnter Number of Points in plane(n):");
    fflush(stdin);
    scanf("%d",&n);
    printf("\nEnter Number of Points inside square(k):");
    fflush(stdin);
    scanf("%d",&k);

   int *h_minArea=NULL;
    
    hipEventCreate(&seq_start);
    hipEventCreate(&seq_stop);

    Point2D *h_points = (Point2D *)malloc(n*sizeof(Point2D));

    if (h_points == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    Point2D *h_sortedX = (Point2D *)malloc(maxWIDTH*sizeof(Point2D));
    if (h_sortedX == NULL)
    {
        fprintf(stderr, "Failed to allocate sorted X host vectors!\n");
        exit(EXIT_FAILURE);
    }
    Point2D *h_sortedY = (Point2D *)malloc(maxWIDTH*sizeof(Point2D));
    if (h_sortedY == NULL)
    {
        fprintf(stderr, "Failed to allocate sorted Y host vectors!\n");
        exit(EXIT_FAILURE);
    }

    int *h_Area = NULL;
    err = hipMalloc((void**)&h_Area,maxWIDTH*maxWIDTH*sizeof(int));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    getRandomInputPoints(n,h_points);

    printf("\nPoints:\n");
    printArray(h_points,n);
    h_minArea = (int*)malloc(sizeof(int));

    Point2D *d_points = NULL;
    err = hipMalloc((void**)&d_points,n*sizeof(Point2D));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_points,h_points,n*sizeof(Point2D),hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    int *d_minArea = NULL;
    err = hipMalloc((void**)&d_minArea,sizeof(int));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_minArea,h_minArea,sizeof(int),hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy minArea from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

     int *d_Area = NULL;
    err = hipMalloc((void**)&d_Area,maxWIDTH*maxWIDTH*sizeof(int));

    if (err != hipSuccess)
    {   
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_Area,h_Area,maxWIDTH*maxWIDTH*sizeof(int),hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {   
        fprintf(stderr, "Failed to copy minArea from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    Point2D *d_sortedX = NULL;
    err = hipMalloc((void**)&d_sortedX,n*sizeof(Point2D));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device sortedX (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    Point2D *d_sortedY = NULL;
    err = hipMalloc((void**)&d_sortedY,n*sizeof(Point2D));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device sortedY (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_sortedX,h_sortedX,n*sizeof(Point2D),hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy sortedX from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_sortedY,h_sortedY,n*sizeof(Point2D),hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy sortedY from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int blocksPerGrid = 3;
    dim3 threadsPerBlock (1024,1,1);
    printf("CUDA kernel launch with %d blocks of 1024 threads\n", blocksPerGrid);
    hipEventRecord(seq_start);
    k_bounding_algorithm<<<blocksPerGrid,threadsPerBlock>>>(d_points,n,k,d_minArea,d_sortedX,d_sortedY,d_Area);
    hipEventRecord(seq_stop);
    hipEventSynchronize(seq_stop);
    float seq_milliseconds = 0;
   err = hipEventElapsedTime(&seq_milliseconds, seq_start, seq_stop);
    if (err != hipSuccess)
   {
        fprintf(stderr, "Failed to calculate elapse time (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    printf("The elapsed time is %.2f ms\n", seq_milliseconds);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_minArea,d_minArea,sizeof(int),hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
     }
    
    printf("\n%d\n",*h_minArea);
    
    err = hipFree(d_minArea);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_minArea (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_points);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_points (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    free(h_points);

    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipFree(d_sortedX);
    hipFree(d_sortedY);
    return 0;
}

