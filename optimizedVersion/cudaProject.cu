#include <stdio.h>
#include<limits.h>
#include <hip/hip_runtime.h>
//#include <thrust/sort.h>
#define maxWIDTH 12

typedef struct
{
int x;
int y;
}Point2D;

__device__ void sortedArea(int *area,int *currentPoint)
{       
        int totalpoint=*currentPoint;
        
        int i,j,temp;
        for(i=0;i<totalpoint-1;i++)
        {       
                for(j=0;j<totalpoint-i-1;j++)
                {       
                        if(area[j]>area[j+1])
                        {       
                                temp=area[j+1];
                                area[j+1]=area[j];
                                area[j]=temp;
                        }
                }
        
        }
}

__device__ int * getAreaWithAxis(Point2D *ptr,int len,int leftEdge,int bottomEdge,int *currentPoint)
{
        int totalpoint=*currentPoint;
        int *area=(int*)malloc(totalpoint * sizeof(int));
        int i=0;
        int ar=0;

        for(i=0;i<totalpoint;i++)
        {
                if((ptr[i].x+1-leftEdge) > (ptr[i].y+1-bottomEdge))
                {
                        ar=(int)(ptr[i].x+1-leftEdge)*((ptr[i].x+1-leftEdge));
                }
                else
                {
                        ar=(int)(ptr[i].y+1-bottomEdge)*((ptr[i].y+1-bottomEdge));
                }
                area[i]=ar;
        }
        return area;
}

__device__ Point2D* getAboveRightPoints(int startx,int starty,Point2D* ptrx,Point2D* ptry,int len,int *currentPoint)
{
    int k=0;
    int i,j;
    int count=0;
    int totalpoint=*currentPoint;
    Point2D* temp;
    int check=0;
    for(i=startx;i<len;i++)
    {
        for(j=starty;j<len;j++)
        {
            if(ptrx[i].x==ptry[j].x && ptrx[i].y==ptry[j].y)
            {
                count++;
                break;
            }
        }
    }

    totalpoint=count;
    temp=(Point2D*)malloc(totalpoint*sizeof(Point2D));
    for(j=starty;j<len;j++)
    {
        for(i=startx;i<len;i++)
        {
            if(ptry[j].x==ptrx[i].x && ptry[j].y==ptrx[i].y)
            {
                check=0;
                for(int m=0;m<k;m++)
                        if(ptry[j].x==temp[m].x && ptry[j].y==temp[m].y)
                                check=1;
                if(check==0)
                {
                        temp[k].x=ptry[j].x;
                        temp[k].y=ptry[j].y;
                        k++;
                        break;
                }
            }
        }
    }
    *currentPoint=k;
    return temp;
}

__device__ void sortingbyAxis(Point2D *points,Point2D *sorted,int n,int x)
{

	int i,j;
	Point2D temp;
        for(i=0;i<n;i++)
        {        
                sorted[i].x=points[i].x;
                sorted[i].y=points[i].y;
        }
        //sort by x co-ordinate
        if(x==1)
        {       
                for(i=0;i<n-1;i++)
                        {
                        for(j=0;j<n-i-1;j++)
                                {       
                                        if(sorted[j].x>sorted[j+1].x)
                                                {       
                                                        temp.x=sorted[j].x;
                                                        temp.y=sorted[j].y;
                                                        sorted[j].x=sorted[j+1].x;
                                                        sorted[j].y=sorted[j+1].y;
                                                        sorted[j+1].x=temp.x;
                                                        sorted[j+1].y=temp.y;
                                                }
                                        }
                                }
        }
        else
        {       
                for(i=0;i<n-1;i++)
                        {
                        for(j=0;j<n-i-1;j++)
                                {       
                                        if(sorted[j].y>sorted[j+1].y)
                                                {       
                                                        temp.x=sorted[j].x;
                                                        temp.y=sorted[j].y;
                                                        sorted[j].x=sorted[j+1].x;
                                                        sorted[j].y=sorted[j+1].y;
                                                        sorted[j+1].x=temp.x;
                                                        sorted[j+1].y=temp.y;
                                                }
                                        }
                                }
        }
}
__global__ void k_bounding_algorithm(Point2D * points,int n,int k,int *finalArea)
{
	int i=threadIdx.x;
	int j=threadIdx.y;


	//Optimization  - Used Shared memory 

	__shared__ Point2D sortedX[maxWIDTH];
	__shared__ Point2D sortedY[maxWIDTH];
	Point2D bottomPoint;
        Point2D leftPoint;
        Point2D *Rpoints=NULL;
        int *area;
	int totalpoints=0;
        
	if(threadIdx.x==0 && threadIdx.y==0)
	{
		sortingbyAxis(points,sortedX,n,1);
		sortingbyAxis(points,sortedY,n,0);
	}
	__syncthreads();
	leftPoint=sortedX[i];
        bottomPoint=sortedY[j];
	if(leftPoint.x <= bottomPoint.x && leftPoint.y >= bottomPoint.y)
	{
            int leftEdge=(int)(leftPoint.x-1);
            int bottomEdge=(int)(bottomPoint.y-1);
            Rpoints=getAboveRightPoints(i,j,sortedX,sortedY,n,&totalpoints);
		if(totalpoints>=k)
                {
			area=getAreaWithAxis(Rpoints,n,leftEdge,bottomEdge,&totalpoints);
            		sortedArea(area,&totalpoints);
			//Optimization - Used atomic operation to find the minimum area
			atomicMin(finalArea,area[k-1]);
		}
	}
}

int main(void)
{
    hipError_t err = hipSuccess;
    hipEvent_t seq_start,seq_stop;    

    int n=12;
    int k=8;
    int *h_minArea=NULL;
    
    hipEventCreate(&seq_start);
    hipEventCreate(&seq_stop);

    Point2D *h_points = (Point2D *)malloc(n*sizeof(Point2D));

    if (h_points == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    h_points[0].x=5;
    h_points[0].y=8;
    h_points[1].x=7;
    h_points[1].y=3;
    h_points[2].x=4;
    h_points[2].y=4;
    h_points[3].x=4;
    h_points[3].y=6;
    h_points[4].x=8;
    h_points[4].y=8;
    h_points[5].x=9;
    h_points[5].y=3;
    h_points[6].x=7;
    h_points[6].y=3;
    h_points[7].x=2;
    h_points[7].y=4;
    h_points[8].x=2;
    h_points[8].y=2;
    h_points[9].x=6;
    h_points[9].y=9;
    h_points[10].x=6;
    h_points[10].y=2;
    h_points[11].x=0;
    h_points[11].y=9;
	
    h_minArea = (int*)malloc(sizeof(int));

    Point2D *d_points = NULL;
    err = hipMalloc((void**)&d_points,n*sizeof(Point2D));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_points,h_points,n*sizeof(Point2D),hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    int *d_minArea = NULL;
    err = hipMalloc((void**)&d_minArea,sizeof(int));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_minArea,h_minArea,sizeof(int),hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy minArea from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int blocksPerGrid = 1;
    dim3 threadsPerBlock (n-k+1,n-k+1,1);
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, (n-k+1)*(n-k+1));
    hipEventRecord(seq_start);
    k_bounding_algorithm<<<blocksPerGrid,threadsPerBlock>>>(d_points,n,k,d_minArea);
    hipEventRecord(seq_stop);
    hipEventSynchronize(seq_stop);
    float seq_milliseconds = 0;
    err = hipEventElapsedTime(&seq_milliseconds, seq_start, seq_stop);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to calculate elapse time (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    printf("The elapsed time is %.2f ms\n", seq_milliseconds);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_minArea,d_minArea,sizeof(int),hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
     }
    
    printf("\nMinimum area of square containing %d points out of %d points is:  %d\n",k,n,*h_minArea);
    
    err = hipFree(d_minArea);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_minArea (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_points);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_points (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    free(h_points);

    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

